#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<vector>
#include<iostream>
#include<string>
#include<sys/time.h>
using namespace std;
int* resultSize;
FILE *fi;
FILE *fp;
struct _INDEX {
	unsigned int  len;
	unsigned int *arr;
} *idx;
struct QueryIndex {
	int size;
	int arr[6];
} *querylist;
int MAXARRS = 2000;
unsigned int i, alen;
unsigned int *aarr;
int j, n;

vector<int> strtoints(char* line) {
	vector<int> arr;
	int i = 0;
	int num = 0;
	while (line[i] == ' ' || (line[i] >= 48 && line[i] <= 57)) {
		num = 0;
		while (line[i] != ' ') {
			num *= 10;
			int tmp = line[i] - 48;
			num += tmp;
			i++;
		}
		i++;
		arr.push_back(num);
	}
	return arr;
}

__global__ void search(QueryIndex * querylist, _INDEX * idx, int QueryNum,int * resultSize) {
	int result[30000];
	int temp[30000];
    int index = threadIdx.x + blockIdx.x * blockDim.x;	//线程编号
    int stride = blockDim.x * gridDim.x;	//跨度
    for (int i = index; i < QueryNum; i+=stride) {
        int TermNum = querylist[i].size;    //query_list[i]表示第i次查询所包含的关键词的集合
		//先把第一个列表作为基准拷贝到临时数组
        int size = 0;
		for (int j = 0;j < idx[querylist[i].arr[0]].len;j++) {
            temp[j] = idx[querylist[i].arr[0]].arr[j];
            size++;
		}
        int tmpsize = 0;
		for (int j = 1;j < TermNum;j++) { //逐个表求交
			for (int k = 0;k < size;k++) {
				//利用二分查找的思想进行元素查找
                bool flag = false;
                int low = 0;
                int high = idx[querylist[i].arr[j]].len - 1;
                int mid = 0;
                while (low <= high) {
                    mid = (low + high) / 2;
                    if (temp[k] == idx[querylist[i].arr[j]].arr[mid]) {
                        flag = true;
                    }
                    else if (temp[k] < idx[querylist[i].arr[j]].arr[mid]) {
                        high = mid - 1;
                    }
                    else {
                        low = mid + 1;
                    }
                }
				if (!flag) {
					//没找到，不添加
				}
				else {
					result[tmpsize] = temp[k];
                    tmpsize++;
				}
			}
			//一次求交之后，result存储临时结果
            for(int k=0;k<tmpsize;k++){
                temp[k]=result[k];
            }
            size = tmpsize;
            tmpsize  = 0;
		}
		//result存储的就是结果，size对应结果集的大小
		resultSize[i] = size;
    }
}

int main() {

	int deviceId;
  int numberOfSMs;

  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
	struct timeval t1, t2;
	double timeuse = 0;
    size_t sizeresult = 1000 * sizeof(int);
	hipMallocManaged(&resultSize, sizeresult);
	hipMemPrefetchAsync(resultSize, sizeresult, deviceId);
    for(int i=0;i<1000;i++){
        resultSize[i]=-1;
    }
	fi = fopen("ExpIndex", "rb");
	if (NULL == fi) {
		printf("Can not open file ExpIndex!\n");
		return 1;
	}
	//idx = (struct _INDEX *)malloc(MAXARRS * sizeof(struct _INDEX));
    size_t sizeidx = MAXARRS * sizeof(struct _INDEX);
	hipMallocManaged(&idx, sizeidx);
	hipMemPrefetchAsync(idx, sizeidx, deviceId);
	if (NULL == idx) {
		printf("Can not malloc %d bytes for idx!\n", MAXARRS * sizeof(struct _INDEX));
		return 2;
	}
	j = 0;
	while (1) {
		fread(&alen, sizeof(unsigned int), 1, fi);
		if (feof(fi)) break;
		//aarr = (unsigned int *)malloc(alen * sizeof(unsigned int));
		size_t sizeaarr = alen * sizeof(unsigned int);
        hipMallocManaged(&aarr, sizeaarr);
		hipMemPrefetchAsync(aarr, sizeaarr, deviceId);
		if (NULL == aarr) {
			printf("Can not malloc %d bytes for aarr!\n", alen * sizeof(unsigned short));
			return 3;
		}
		for (int i = 0;i < alen;i++) {
			fread(&aarr[i], sizeof(unsigned int), 1, fi);
			if (feof(fi)) break;
		}
		if (feof(fi)) break;
		idx[j].len = alen;
		idx[j].arr = aarr;
		j++;
		if (j >= MAXARRS) {
			printf("Too many arrays(>=%d)!\n", MAXARRS);
			break;
		}
	}
	fclose(fi);

	//现在已经有一个idx数组存储了这个倒排索引文件，idx[i].arr表示第i个关键词的倒排索引链表
	//下面是query_list代表查询的二维数组，大概能到2000个关键词，所以上面的max可以设置为2000

	fp = fopen("ExpQuery", "r");
	vector<vector<int> > query_list;

	vector<int> arr;
	char* line = new char[100];
	while ((fgets(line, 100, fp)) != NULL)
	{
		arr = strtoints(line);
		query_list.push_back(arr);
	}
	fclose(fp);
    size_t sizeqlist = 1005 * sizeof(struct QueryIndex);
	hipMallocManaged(&querylist, sizeqlist);
	hipMemPrefetchAsync(querylist, sizeqlist, deviceId);
	for (int i = 0;i < query_list.size();i++) {
		querylist[i].size = query_list[i].size();
		for (int j = 0;j < query_list[i].size();j++) {
			querylist[i].arr[j] = query_list[i][j];
		}
	}

	//实现按表求交的平凡算法
	int QueryNum = 10; //代表要处理的查询次数
    gettimeofday(&t1, NULL);
	int threadnum = 1;
    int blocknum = numberOfSMs * 32;
	search<<<blocknum,threadnum>>>(querylist, idx, QueryNum,resultSize); 
	hipDeviceSynchronize();
    gettimeofday(&t2, NULL);
	timeuse += (t2.tv_sec-t1.tv_sec) * 1000000 + t2.tv_usec-t1.tv_usec;
	cout << "time_use=" << timeuse << endl;
    for(int i=0;i<QueryNum;i++){
        cout<<resultSize[i]<<endl;
    }
	for (j = 0;j < n;j++) free(idx[j].arr);
	hipFree(idx);
    hipFree(querylist);
    hipFree(resultSize);
	hipFree(aarr);
	return 0;
}