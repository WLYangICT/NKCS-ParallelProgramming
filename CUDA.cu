
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<vector>
#include<iostream>
#include<string>
#include<sys/time.h>
using namespace std;
vector<vector<unsigned int> > results;
int* resultSize;
FILE *fi;
FILE *fp;
struct _INDEX {
	unsigned int  len;
	unsigned int *arr;
} *idx;
struct QueryIndex {
	int size;
	int arr[6];
} *querylist;
int MAXARRS = 2000;
unsigned int i, alen;
unsigned int *aarr;
int j, n;

vector<int> strtoints(char* line) {
	vector<int> arr;
	int i = 0;
	int num = 0;
	while (line[i] == ' ' || (line[i] >= 48 && line[i] <= 57)) {
		num = 0;
		while (line[i] != ' ') {
			num *= 10;
			int tmp = line[i] - 48;
			num += tmp;
			i++;
		}
		i++;
		arr.push_back(num);
	}
	return arr;
}

bool find(unsigned int e, _INDEX list) { //进行二分查找
	int low = 0;
	int high = list.len - 1;
	int mid = 0;
	while (low <= high) {
		mid = (low + high) / 2;
		if (e == list.arr[mid]) {
			return true;
		}
		else if (e < list.arr[mid]) {
			high = mid - 1;
		}
		else {
			low = mid + 1;
		}
	}
	return false;
}

int result[100000];
int temp[100000];

__global__ void search(QueryIndex * querylist, _INDEX * idx, int QueryNum,int * resultSize) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int result[100000];
    int temp[100000];
    for (int i = index; i < QueryNum; i+=stride) {
		resultSize[i] = idx[querylist[0].arr[0]].arr[0];
        int TermNum = querylist[i].size;    //query_list[i]表示第i次查询所包含的关键词的集合
		//先把第一个列表作为基准拷贝到临时数组
        int size = 0;
		for (int j = 0;j < idx[querylist[i].arr[0]].len;j++) {
            temp[j] = idx[querylist[i].arr[0]].arr[j];
            size++;
		}
        int tmpsize = 0;
		for (int j = 1;j < TermNum;j++) { //逐个表求交
			for (int k = 0;k < size;k++) {
                bool flag = false;
                int low = 0;
                int high = idx[querylist[i].arr[j]].len - 1;
                int mid = 0;
                while (low <= high) {
                    mid = (low + high) / 2;
                    if (temp[k] == idx[querylist[i].arr[j]].arr[mid]) {
                        flag = true;
                    }
                    else if (temp[k] < idx[querylist[i].arr[j]].arr[mid]) {
                        high = mid - 1;
                    }
                    else {
                        low = mid + 1;
                    }
                }
				if (!flag) {
					//没找到，不添加
				}
				else {
					result[tmpsize] = temp[k];
                    tmpsize++;
				}
			}
			//一次求交之后，result存储临时结果
            for(int k=0;k<tmpsize;k++){
                temp[k]=result[k];
            }
            size = tmpsize;
            tmpsize  = 0;
		}
		resultSize[i] = size;
    }
}

int main() {
struct timeval t1, t2;
double timeuse = 0;
    size_t sizeresult = 1000 * sizeof(int);
	hipMallocManaged(&resultSize, sizeresult);
    for(int i=0;i<1000;i++){
        resultSize[i]=9999;
    }
	fi = fopen("ExpIndex", "rb");
	if (NULL == fi) {
		printf("Can not open file ExpIndex!\n");
		return 1;
	}
	//idx = (struct _INDEX *)malloc(MAXARRS * sizeof(struct _INDEX));
    size_t sizeidx = MAXARRS * sizeof(struct _INDEX);
	hipMallocManaged(&idx, sizeidx);
	if (NULL == idx) {
		printf("Can not malloc %d bytes for idx!\n", MAXARRS * sizeof(struct _INDEX));
		return 2;
	}
	j = 0;
	while (1) {
		fread(&alen, sizeof(unsigned int), 1, fi);
		if (feof(fi)) break;
		//aarr = (unsigned int *)malloc(alen * sizeof(unsigned int));
		size_t sizeaarr = alen * sizeof(unsigned int);
        hipMallocManaged(&aarr, sizeaarr);
		if (NULL == aarr) {
			printf("Can not malloc %d bytes for aarr!\n", alen * sizeof(unsigned short));
			return 3;
		}
		for (int i = 0;i < alen;i++) {
			fread(&aarr[i], sizeof(unsigned int), 1, fi);
			if (feof(fi)) break;
		}
		if (feof(fi)) break;
		idx[j].len = alen;
		idx[j].arr = aarr;
		j++;
		if (j >= MAXARRS) {
			printf("Too many arrays(>=%d)!\n", MAXARRS);
			break;
		}
	}
	fclose(fi);

	//现在已经有一个idx数组存储了这个倒排索引文件，idx[i].arr表示第i个关键词的倒排索引链表
	//下面是query_list代表查询的二维数组，大概能到2000个关键词，所以上面的max可以设置为2000

	fp = fopen("ExpQuery", "r");
	vector<vector<int> > query_list;

	vector<int> arr;
	char* line = new char[100];
	while ((fgets(line, 100, fp)) != NULL)
	{
		arr = strtoints(line);
		query_list.push_back(arr);
	}
	fclose(fp);
    size_t sizeqlist = 1005 * sizeof(struct QueryIndex);
	hipMallocManaged(&querylist, sizeqlist);
	for (int i = 0;i < query_list.size();i++) {
		querylist[i].size = query_list[i].size();
		for (int j = 0;j < query_list[i].size();j++) {
			querylist[i].arr[j] = query_list[i][j];
		}
	}
	//实现按表求交的平凡算法
	int QueryNum = 100; //代表要处理的查询次数
    gettimeofday(&t1, NULL);
	int threadnum = 128;
    int blocknum = 10;
	search<<<blocknum,threadnum>>>(querylist, idx, QueryNum,resultSize); // compute interbody forces
    //test<<<blocknum,threadnum>>>(resultSize,QueryNum,idx,querylist);
	hipDeviceSynchronize();
    //cout<<querylist[0].arr[0]<<endl;
    //cout<<"test:"<<(long long)idx[1116].arr<<endl;
    gettimeofday(&t2, NULL);
	timeuse += (t2.tv_sec-t1.tv_sec) * 1000000 + t2.tv_usec-t1.tv_usec;
	cout << "time_use=" << timeuse << endl;
    for(int i=0;i<QueryNum;i++){
        cout<<resultSize[i]<<endl;
    }
	for (j = 0;j < n;j++) free(idx[j].arr);
	hipFree(idx);
    hipFree(querylist);
    hipFree(resultSize);
	hipFree(aarr);
	return 0;
}